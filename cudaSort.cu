
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cstring>
#include <algorithm>
#include <vector>
#include <chrono>

#define MAX_VALUE 1000 // maksymalna wartość, jaką może przyjąć element tablicy

// sortowanie przysto-nieparzyste na CUDA
__global__ void odd_even_sort(int *a, int n)
{
    int phase, temp;

    // iteracja po fazach
    for (phase = 0; phase < n; phase++)
    {
        // faza parzysta
        if (phase % 2 == 0)
        {
            // indeks biezacego bloku w watku
            // kazdy watek przetwarza tylko jeden element tablicy
            if (threadIdx.x % 2 == 0 && threadIdx.x < n - 1)
            {
                if (a[threadIdx.x] > a[threadIdx.x + 1])
                {
                    temp = a[threadIdx.x];
                    a[threadIdx.x] = a[threadIdx.x + 1];
                    a[threadIdx.x + 1] = temp;
                }
            }
        }
        // faza nieparzysta
        else
        {
            if (threadIdx.x % 2 != 0 && threadIdx.x < n - 1)
            {
                if (a[threadIdx.x] > a[threadIdx.x + 1])
                {
                    temp = a[threadIdx.x];
                    a[threadIdx.x] = a[threadIdx.x + 1];
                    a[threadIdx.x + 1] = temp;
                }
            }
        }
        __syncthreads();
    }
}

// sprawdzenie, czy tablica została poprawnie posortowana
void self_test(int *initial, int *sorted, int length)
{
    std::vector<int> initial_copy(initial, initial + length);
    std::sort(initial_copy.begin(), initial_copy.end());

    for (int i = 0; i < length; i++)
    {
        if (initial_copy[i] != sorted[i])
        {
            std::cout << "[Error] Array is not sorted" << std::endl;
            return;
        }
    }
    std::cout << "[OK] Array is sorted" << std::endl;
}

// generowanie tablicy losowych liczb
void generate_array(int *a, int size)
{
    std::srand(std::time(nullptr));

    for (int i = 0; i < size; i++)
    {
        a[i] = std::rand() % MAX_VALUE;
    }
}

// wypisanie tablicy
void print_array(int *a, int size)
{
    for (int i = 0; i < size; i++)
    {
        std::cout << a[i] << std::endl;
    }
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        std::cout << "Error: Please provide the size of the array to sort and the number of threads to use" << std::endl;
        return -1;
    }
    // konwersja argumentów string na int
    int size = std::atoi(argv[1]);
    int threads = std::atoi(argv[2]);

    // stworzenie dynamicznej tablicy
    int *a = new int[size];
    int *initial = new int[size];

    generate_array(a, size);

    // skopiowanie tablicy 'a' do tablicy 'initial'
    std::memcpy(initial, a, size * sizeof(int));

    //printf("\nUnsorted array:\n");
    //print_array(a, size);

    // Alokacja pamięci na GPU
    int *dev_a;
    hipMalloc((void **)&dev_a, size * sizeof(int));

    // Kopiowanie danych z CPU do GPU
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    // Uruchamianie sortowania przysto-nieparzystego na GPU
    odd_even_sort<<<1, threads>>>(dev_a, size);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;
    printf("\nTime: %f\n", diff.count());

    // Kopiowanie wyników z GPU do CPU
    hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);

    //printf("\nSorted array:\n");
    //print_array(a, size);
    self_test(initial, a, size);

    // Zwolnienie pamięci na GPU
    hipFree(dev_a);

    // Zwolnienie pamięci na CPU
    delete[] a;
    delete[] initial;

    return 0;
}